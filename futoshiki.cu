
#include <hip/hip_runtime.h>
#include<iostream>
#include<string>
#include<sstream>
#include<fstream>
#include <cstdio>
#include <vector>
#include <set>

#define SIZE 5 //Matrix size
#define INPUTSIZE 2306451

//THIS IS how many constraints per instance (at max): 60
//THIS IS how many grids: 144000

__device__ bool FindUnassignedLocation(int* matrix, int &row, int &col)
{
    for (row = 0; row < 5; row++)
        for (col = 0; col < 5; col++)
            if (matrix[row * 5 + col] == -2)
                return true;
    return false;
}


__device__ bool isSafe(int* matrix, int row, int col, int num, int * constraints, int constraint_size)
{
    for (int row = 0; row < 5; row++)
        if (matrix[row * 5 + col] == num)
            return false;


    for (int col = 0; col < 5; col++)
        if (matrix[row * 5 + col] == num)
            return false;


    for(long unsigned int i = 0; i < constraint_size; i+=4)
    {
        if(row == constraints[i] && col == constraints[i+1] && matrix[constraints[i+2] * 5 + constraints[i+3]] != -2 && num < matrix[constraints[i+2] * 5 + constraints[i+3]])
            return false;
        else if(row == constraints[i+2] && col == constraints[i+3] && matrix[constraints[i] * 5 + constraints[i+1]] != -2 && matrix[constraints[i] * 5 + constraints[i+1]] < num)
            return false;
    }
    return true;
}

__global__ void GPU_Futoshiki(int* grids, int* constraints, int* constraint_sizes, int* constraint_beginnings)
{

    int tid = threadIdx.x;
    int x_block = blockIdx.x;
    
    volatile __shared__ bool not_found_flag;  // shared flag to stop other threads when solution found
    __shared__ int local_constraints[60];  // this is a dummy size, large enough to fit for all cases
    __shared__ int constraint_size;  // to get how many constraints do we have (item-wise)
    __shared__ int constraint_start;  // to get where we should start from
    __shared__ int local_final[25];  // for storing the final results locally (also used as a temporary storage)

    // initialization of the shared variables
    not_found_flag = true;
    constraint_size = constraint_sizes[x_block] * 4;
    constraint_start = constraint_beginnings[x_block] * 4;

    if(tid < constraint_size)
        local_constraints[tid] = constraints[constraint_start + tid];
        /*
        if(blockIdx.x == 3 && tid == 0)
        {
            for(int yy = 0; yy < constraint_size; yy+=4)
            {
                printf("%d, %d, %d, %d\n", local_constraints[yy]+1, local_constraints[yy+1]+1, local_constraints[yy+2]+1, local_constraints[yy+3]+1);
            }
        }
        */

    
    if(tid < 25)  // we will assign 5 thread to each cell, 1 thread per value in a cell
    {
        local_final[tid] = grids[x_block * 25 + tid];  // saving the grid into shared memory first (local_final used as temporary storage)

        __syncthreads();
    }
    if(tid < 5)  // we will assign 5 thread to each cell, 1 thread per value in a cell
    {
        int futoshiki[25];  // copy per thread

        for(int q = 0; q < 25; q++)  // filling the copy per thread from shared mem
            futoshiki[q] = local_final[q];  // :)
        
        // start solving the futoshiki
        if(futoshiki[tid] == -2)  // if that cell is empty
        {
            int row;
            int col;

            //futoshiki[row * 5 + col] = value;  // if so, change the value of the current cell
            int staque[50];  // stack implementation via int list
            int stack_counter = 0;  // need for stack implementation, consider as bookmark xd
            int toStack;  // value to store what's popped from the stack
            //int kaputt = 0;  // control variable for if there is no solution
            int allowed_value = 0;  // for roll-back mechanism, allowing us to remember which value to try next
            bool dead_end;  // if we need to roll-back or not
            
            while(not_found_flag)
            {
                if(!FindUnassignedLocation(futoshiki, row, col))
                {
                    not_found_flag = false;
                    for(int x = 0; x < 25; x++)
                        local_final[x] = futoshiki[x];
                }
                else
                {
                    //printf("RETURNED row: %d, col: %d\n", row, col);
                    dead_end = true;  // dummy initialization, if it does not turn to false in the for loop below, it means we have a problem :)
                    //printf("banned value: %d,  deadend: %d\n", banned_value, dead_end);
                    for (int num = allowed_value; num < 5 && dead_end; num++)  // try the values
                    {
                        if (isSafe(futoshiki, row, col, num, local_constraints, constraint_size))  // if the value fits, put it
                        {
                            //printf("now chainging row: %d, col: %d, with num: %d\n", row, col, num);
                            dead_end = false;  // set the dead end to false, since we found a new value
                            allowed_value = 0;  // reset this, since we found a new value
                            futoshiki[row * 5 + col] = num;  // a new step in the matrix 
                            toStack = row * 100 + col * 10 + num;
                            staque[stack_counter++] = toStack;  // push this cell and it's value into the stack, in case of we screw things up XD
                        }
                    }
                    if(dead_end) // means we have a dead end in this cell, so we need to roll back
                    {  // by popping once from our stack
                        toStack = staque[--stack_counter];  // pop the stack and store the value
                        allowed_value = toStack % 10 + 1;  // we need to add 1, otherwise we will be trying the same value over and over again
                        col = (toStack/10) % 10;  // get the cell col
                        row = toStack / 100;  // get the cell row
                        futoshiki[row * 5 + col] = -2;  // set this cell to uninitialized again
                    }
                }
            }
            
        }
        __syncthreads();  // wait for all other threads
    }
    if(not_found_flag == false)
        if(tid < 25) // this is put for debugging, remove and merge with above if when debug done
            grids[x_block * 25 + tid] = local_final[tid];  // write the result back to the global gpu memory
}


int main(int argc, char** argv)
{
  
    std::string filename(argv[1]);
    std::ifstream file(filename.c_str());
    std::ifstream scout(filename.c_str());
    
    int no_grids;
    file >> no_grids;

    int dummy;
    scout >> dummy;

    int* grids = new int[no_grids * 25];
    

    int elem0, elem1, elem2, elem3, elem4;
    int pre_cursor = 0;
    int cursor = 0;
    int csize = 0;
    
    std::string file_line;
    std::string scout_line;

    int* constraint_sizes = new int[no_grids];
    int* constraint_beginnings = new int[no_grids];
    
    std::getline(scout, scout_line);//These are for spare lines
    std::getline(scout, scout_line);
    for(int i = 0; i < INPUTSIZE; i++)
    {
        std::getline(scout, scout_line);
        if(scout_line == "-------")
        {
            csize = i - pre_cursor - 5;
            constraint_sizes[cursor] = csize;
            cursor++;
            pre_cursor = i+1;
        }
    }

    int sum = 0;
    int temp_size;

    std::vector<int> constraint_vector;  // we need a dynamic one that can expand
    std::set<int> constraint_set;  // there are multiple copies of the constraints, WHY ARE you torturing us :(

    std::getline(file, file_line);
    for(int i = 0; i < no_grids; i++)
    {
        std::getline(file, file_line);
        for(int j = 0; j < SIZE; j++)
        {
            std::getline(file, file_line);
            std::istringstream iss(file_line);
            iss >> elem0 >> elem1 >> elem2 >> elem3 >> elem4;
            grids[i*25 + j*5 + 0] = elem0 - 1;
            grids[i*25 + j*5 + 1] = elem1 - 1;
            grids[i*25 + j*5 + 2] = elem2 - 1;
            grids[i*25 + j*5 + 3] = elem3 - 1;
            grids[i*25 + j*5 + 4] = elem4 - 1;
        }
        for(int c = 0; c < constraint_sizes[i]; c++)
        {
            std::getline(file, file_line);
            std::istringstream iss(file_line);
            iss >> elem0 >> elem1 >> elem2 >> elem3;
            elem4 = elem0 * 1000 + elem1 * 100 + elem2 * 10 + elem3;
            constraint_set.insert(elem4);
        } 
        temp_size = constraint_set.size();
        constraint_sizes[i] = temp_size;
        constraint_beginnings[i] = sum;
        sum += temp_size;
        
        
        for (std::set<int>::iterator it=constraint_set.begin(); it!=constraint_set.end(); ++it)
        {
            elem4 = *it;
            constraint_vector.push_back(elem4 / 1000 - 1);  // 4th digit
            constraint_vector.push_back((elem4 / 100) % 10 - 1);  // 3rd digit
            constraint_vector.push_back((elem4 / 10) % 10 - 1);  // 2nd digit
            constraint_vector.push_back(elem4 % 10 - 1);  // 1st digit
        }
        constraint_set.clear();

    }


    temp_size = constraint_vector.size();
    int * constraints = new int[temp_size];
    elem4 = 0;  // my favourite dummy int :)
    for (std::vector<int>::iterator it = constraint_vector.begin(); it != constraint_vector.end(); it++)
    {
        constraints[elem4++] = *it;
    }

    int *grids_d, *constraints_d, *constraint_sizes_d, *constraint_beginnings_d;

    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    //YOUR MEMORY OPERATIONS//Time accordingly
    hipEventRecord(start, 0);
    hipMalloc((void**)&grids_d, no_grids * 25 * sizeof(int));
    hipMemcpy(grids_d, grids, no_grids * 25 * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&constraints_d, temp_size * sizeof(int));
    hipMemcpy(constraints_d, constraints, temp_size * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&constraint_sizes_d, no_grids * sizeof(int));
    hipMemcpy(constraint_sizes_d, constraint_sizes, no_grids * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**)&constraint_beginnings_d, no_grids * sizeof(int));
    hipMemcpy(constraint_beginnings_d, constraint_beginnings, no_grids * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GPU Memory preparation duration: %f ms \n", time);
    //YOUR MEMORY OPERATIONS//
    

    //KERNEL CALL//Time accordingly
    hipEventRecord(start, 0);
    GPU_Futoshiki<<<no_grids, 96>>>(grids_d, constraints_d, constraint_sizes_d, constraint_beginnings_d);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Kernel Duration: %f ms \n", time);
    //KERNEL CALL//


    //YOUR MEMORY OPERARIONS//Time accordingly
    hipEventRecord(start, 0);
    hipMemcpy(grids, grids_d, no_grids * 25 * sizeof(int), hipMemcpyDeviceToHost); // copy the result back to CPU
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("GPU to CPU Data Transfer Duration: %f ms \n", time);
    //YOUR MEMORY OPERARIONS//



    // free cuda mem
    hipFree(grids_d);
    hipFree(constraints_d);
    hipFree(constraint_sizes_d);
    hipFree(constraint_beginnings_d);
  
    //OUTPUT FILE
    std::ofstream myfile;
    myfile.open("solution.txt");
    myfile << no_grids << "\n" << "-------" << "\n";
    for(int i = 0; i < no_grids; i++)
    {
        for(int j = 0; j < SIZE; j++)
        {
            elem0 = grids[i*25 + j*5 + 0] + 1;
            elem1 = grids[i*25 + j*5 + 1] + 1;
            elem2 = grids[i*25 + j*5 + 2] + 1;
            elem3 = grids[i*25 + j*5 + 3] + 1;
            elem4 = grids[i*25 + j*5 + 4] + 1;
            myfile << elem0 << " " << elem1 << " " << elem2 << " " << elem3 << " " << elem4 << "\n";
        }
        myfile << "-------" << "\n";
    }
    myfile.close();
    //OUTPUT FILE

    
    /*
    cudaFree(grids_d);
    cudaFree(constraints_d);
    cudaFree(constraint_sizes_d);
    cudaFree(constraint_beginnings_d);
    */
    //Deallocate
    delete[] grids;
   
    delete[] constraints;
  
    delete[] constraint_sizes;

    delete[] constraint_beginnings;
}
